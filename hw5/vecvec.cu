
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void innerprod(double* ip_ptr, const double* a, const double* b, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i]*b[i];
  *ip_ptr = sum;
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

// Warp divergence
__global__ void reduction_kernel0(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x %   2 == 0) smem[threadIdx.x] += smem[threadIdx.x + 1];
  __syncthreads();
  if (threadIdx.x %   4 == 0) smem[threadIdx.x] += smem[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x %   8 == 0) smem[threadIdx.x] += smem[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x %  16 == 0) smem[threadIdx.x] += smem[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x %  32 == 0) smem[threadIdx.x] += smem[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x %  64 == 0) smem[threadIdx.x] += smem[threadIdx.x + 32];
  __syncthreads();
  if (threadIdx.x % 128 == 0) smem[threadIdx.x] += smem[threadIdx.x + 64];
  __syncthreads();
  if (threadIdx.x % 256 == 0) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x % 512 == 0) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[threadIdx.x] + smem[threadIdx.x + 512];
}

// Shared memory bank conflicts
__global__ void reduction_kernel1(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x *   2] += smem[threadIdx.x *   2 +   1];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x *   4] += smem[threadIdx.x *   4 +   2];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x *   8] += smem[threadIdx.x *   8 +   4];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x *  16] += smem[threadIdx.x *  16 +   8];
  __syncthreads();
  if (threadIdx.x <  32) smem[threadIdx.x *  32] += smem[threadIdx.x *  32 +  16];
  __syncwarp();
  if (threadIdx.x <  16) smem[threadIdx.x *  64] += smem[threadIdx.x *  64 +  32];
  __syncwarp();
  if (threadIdx.x <   8) smem[threadIdx.x * 128] += smem[threadIdx.x * 128 +  64];
  __syncwarp();
  if (threadIdx.x <   4) smem[threadIdx.x * 256] += smem[threadIdx.x * 256 + 128];
  __syncwarp();
  if (threadIdx.x <   2) smem[threadIdx.x * 512] += smem[threadIdx.x * 512 + 256];
  __syncwarp();
  if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[512];
}

__global__ void innerprod_kernel2(double* sum, const double* a, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx]*b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void reduction_kernel2(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}
int main() {
  long N = (1UL<<25);
  printf("N=%d\n", N);
  double *x;
  double *y; // we want to compute the inner product of x and y
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&y, N * sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) x[i] = 1.0/(i+1);
  for (long i = 0; i < N; i++) y[i] = 1.0/(i+1);

  double ip_ref, sum;
  double tt = omp_get_wtime();
  innerprod(&ip_ref, x, y, N);
  printf("CPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *y_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&y_d, N*sizeof(double));
  long N_work = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  hipMalloc(&z_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks

  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(y_d, y, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();


  double* sum_d = z_d;
  long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
  innerprod_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d, x_d,y_d, N);
  while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel2<<<Nb,BLOCK_SIZE>>>(sum_d + N, sum_d, N);
    sum_d += N;
  }


  hipMemcpyAsync(&sum, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(sum-ip_ref));

  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipHostFree(x);
  hipHostFree(y);
  return 0;
}

